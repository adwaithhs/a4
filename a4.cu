
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <getopt.h>
#include <sys/stat.h>
#include <sys/types.h>

#define KEY(x) *((uint32_t*)x)

int comp32(const void *elem1, const void *elem2) 
{
    uint32_t f = KEY(elem1);
    uint32_t s = KEY(elem2);
    if (f > s) return  1;
    if (f < s) return -1;
    return 0;
}

void print_array(uint64_t *arr, uint64_t n) {
    for (int i = 0; i < n; i++) {
        printf("%" PRIu32 " ", KEY(arr[i]));
    }
    printf("\n");
}

uint64_t* cpu_sort(uint64_t* data, uint64_t n, int p) {
    if (n <= 4*p*p)
    {
        qsort(data, n, 8, comp32);
        return data;
    }
    uint64_t q = n / p;
    int r = n % p;
    
    uint64_t *R = (uint64_t*)malloc(p*p * sizeof(*R));
    uint64_t *S = (uint64_t*)malloc((p-1) * sizeof(*S));
    uint64_t *m = (uint64_t*)malloc(p * sizeof(*m));
    uint64_t *c = (uint64_t*)malloc(p * sizeof(*c));
    uint64_t *h = (uint64_t*)malloc(p * sizeof(*h));
    uint64_t *final = (uint64_t*)malloc(n * sizeof(*final));
    
    #pragma omp parallel
    {
        #pragma omp single
        {
            uint64_t a = 0;
            for (int i = 0; i < p; i++)
            {
                uint64_t size;
                if (i < r)
                {
                    size = q+1;
                }
                else
                {
                    size = q;
                }
                #pragma omp task
                qsort(data + a, size, 8, comp32);
                a += size;
            }
            #pragma omp taskwait
            int k = 0;
            a = 0;
            for (int i = 0; i < p; i++)
            {
                uint64_t size;
                if (i < r)
                {
                    size = q+1;
                }
                else
                {
                    size = q;
                }
                

                for (uint64_t j = a; j < a + size; j+= size/p)
                {
                    R[k] = data[j];
                    k++;
                }
                
                a += size;
            }

            qsort(R, p*p, 8, comp32);
            for (int j = 0; j < p-1; j++)
            {
                S[j] = R[(j+1)*p];
            }
            for (int j = 0; j < p; j++)
            {
                #pragma omp task
                {
                    m[j] = 0;
                    for (uint64_t i = 0; i < n; i++)
                    {
                        if ((j == 0 || S[j-1] < data[i]) && (j == p-1 || data[i] <= S[j]))
                        {
                            m[j]++;
                        }
                    }
                }
            }
            #pragma omp taskwait
            c[0] = 0;
            h[0] = 0;
            for (int i = 1; i < p; i++)
            {
                c[i] = c[i-1] + m[i-1];
                h[i] = c[i];
            }

            for (int j = 0; j < p; j++)
            {
                #pragma omp task
                {
                    uint64_t k = 0;
                    for (uint64_t i = 0; i < n; i++)
                    {
                        uint32_t key = KEY(data[i]);
                        if ((j == 0 || S[j-1] < key) && (j == p-1 || key <= S[j]))
                        {
                            final[c[j]+k] = data[i];
                        }
                    }
                }
            }
            #pragma omp taskwait

            for (int j = 0; j < p; j++)
            {
                #pragma omp task 
                qsort(final + c[j], m[j], 8, comp32);
            }
            #pragma omp taskwait
        }
    }
    return final;
}

__global__ void step(uint64_t *ddata, int j, int k)
{
    int i, l;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    l = i^j;

    if (l > i) {
        if ((i&k)==0) {
            if (KEY(ddata[i]) > KEY(ddata[l])) {
                uint64_t temp = ddata[i];
                ddata[i] = ddata[l];
                ddata[l] = temp;
            }
        }
        if ((i&k)!=0) {
            if (KEY(ddata[i]) < KEY(ddata[l])) {
                uint64_t temp = ddata[i];
                ddata[i] = ddata[l];
                ddata[l] = temp;
            }
        }
    }
}


void gpu_sort(uint64_t* data, uint64_t n, uint64_t blocks, uint64_t threads) {
    uint64_t *ddata;

    hipMalloc(&ddata, n*sizeof(uint64_t));
    hipMemcpy(ddata, data, n*sizeof(uint64_t), hipMemcpyHostToDevice);

    int j, k;
    for (k = 2; k <= n; k <<= 1) {
        for (j=k>>1; j>0; j=j>>1) {
            step<<<blocks, threads>>>(ddata, j, k);
        }
    }
    hipMemcpy(data, ddata, n*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(ddata);
}

uint64_t * merge(uint64_t* a, uint64_t n, uint64_t* b, uint64_t m) {
    uint64_t *c = (uint64_t*)malloc((n*m) * sizeof(*c));
    uint64_t i = 0, j = 0, k = 0;
    
    while (i < n && j < m) {
        if (a[i] < b[j]) {
            c[k] = a[i];
            i++;
        } else {
            c[k] = b[j];
            j++;
        }
        k++;
    }

    while (i < n) {
        c[k] = a[i];
        i++;
        k++;
    }

    while (j < m) {
        c[k] = b[j];
        j++;
        k++;
    }
    return c;
}

int main(int argc, char** argv) {
    static struct option long_options[] = {
        {"inputname", required_argument, 0, 'i'},
        {"outputpath", required_argument, 0, 'o'},
        {"p", required_argument, 0, 'p'},
        {"gpu", no_argument, 0, 'g'},
        {0, 0, 0, 0}
    };
    int p, use_gpu = 0;
    char _i[] = "input";
    char _o[] = "output";
    char *input_path = _i;
    char *output_path = _o;
    while (1) {
        int c = getopt_long(argc, argv, "", long_options, NULL);
        if (c == -1) break;
        switch (c)
        {
        case 'i':
            input_path = (char*)malloc((strlen(optarg)+1)*sizeof(char));
            strcpy(input_path, optarg);
            break;
        case 'o':
            output_path = (char*)malloc((strlen(optarg)+1)*sizeof(char));
            strcpy(output_path, optarg);
            break;
        case 'p':
            p = atoi(optarg);
            break;
        case 'g':
            use_gpu = 1;
            break;
        
        default:
            break;
        }
    }

    
    uint64_t* data;

    FILE *fs = fopen(input_path, "rb");
    fseeko(fs, 0, SEEK_END);
    off_t n = ftello(fs) / 8;


    fseeko(fs, 0, SEEK_SET);
    data = (uint64_t*)malloc(n*8);
    fread(data, 8, n, fs);
    fclose(fs);
    
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);

    if (n <= 4*p*p)
    {
        qsort(data, n, 8, comp32);
        FILE *fs = fopen(output_path, "wb");
        fwrite(data, 8, n, fs);
        fclose(fs);
        return;
    }

    if (use_gpu == 0 || nDevices == 0) {
        uint64_t* final = cpu_sort(data, n, p);
        FILE *fs = fopen(output_path, "wb");
        fwrite(final, 8, n, fs);
        fclose(fs);
        return;
    }
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int threads = prop.maxThreadsPerBlock;

    uint64_t r = 1;
    while (2*r < n) {
        r = 2*r;
    }
    if (r==n) {
        if (r <= threads) {
            gpu_sort(data, n, 1, n);
        } else {
            gpu_sort(data, n, n/threads, threads);
        }
        FILE *fs = fopen(output_path, "wb");
        fwrite(data, 8, n, fs);
        fclose(fs);
        return;
    }
    int b, t;
    if (r <= threads) {
        b = 1;
        t = r;
    } else {
        b = r/threads;
        t = threads;
    }

    uint64_t *ddata;

    hipMalloc(&ddata, n*sizeof(uint64_t));
    hipMemcpy(ddata, data, n*sizeof(uint64_t), hipMemcpyHostToDevice);

    int j, k;
    for (k = 2; k <= n; k <<= 1) {
        for (j=k>>1; j>0; j=j>>1) {
            step<<<b, t>>>(ddata, j, k);
        }
    }
    uint64_t* final = cpu_sort(data + r, n - r, p);
    hipMemcpy(data, ddata, n*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(ddata);

    final = merge(data, r, final, n-r);
    fs = fopen(output_path, "wb");
    fwrite(final, 8, n, fs);
    fclose(fs);

    return 0;
}
